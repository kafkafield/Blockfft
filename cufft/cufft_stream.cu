#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include ""
//#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#include <ctime>
#include <time.h>
#include <stdio.h>
#include <iostream>
#include <math.h>
#include <hipfft/hipfft.h>
#include <fstream>

using namespace std;
typedef float2 Complex;

int main()
{
	int test_stream = 8;
	int test_number = 10000;
	for (int iter = 13; iter <= 13; ++iter)
	{
		for (int i_stream = 8; i_stream <= test_stream; ++i_stream)
		{
			int N = iter;
			int SIZE = N*N;
			
			hipStream_t streams[8];
			for (int i = 0; i < i_stream; ++i)
				hipStreamCreate(&streams[i]);

			Complex *fg[8];
			for (int i = 0; i < i_stream; ++i)
				fg[i] = new Complex[SIZE];
			for (int j = 0; j < i_stream; ++j)
				for (int i = 0; i < SIZE; i++){
					fg[j][i].x = 1; 
					fg[j][i].y = 0;
				}
    
			int mem_size = sizeof(Complex)* SIZE;

			Complex *d_signal[8];
			for (int i = 0; i < i_stream; ++i) {
				//cout << i << endl;
				checkCudaErrors(hipMalloc((void **) &(d_signal[i]), mem_size)); 
				checkCudaErrors(hipMemcpyAsync(d_signal[i], fg[i], mem_size, 
					hipMemcpyHostToDevice, streams[i]));
			}
	
			hipDeviceSynchronize();
			for (int i = 0; i < i_stream; ++i) 
				hipStreamSynchronize(streams[i]);
	
			// CUFFT plan
			hipfftHandle plan[8];	
			for (int i = 0; i < i_stream; ++i) {
				hipfftPlan2d(&plan[i], N, N, HIPFFT_C2C);
				hipfftSetStream(plan[i], streams[i]);
			}
	
			// Transform signal and filter
			clock_t start, end;
			start = clock();
			for (int j = 0; j < test_number / i_stream; ++j) {
				for (int i = 0; i < i_stream; ++i) {
					hipfftExecC2C(plan[i], (hipfftComplex *)d_signal, (hipfftComplex *)d_signal, 
						HIPFFT_FORWARD);
				}
				hipDeviceSynchronize();
			}
			end = clock();
			double fft_time = (double)(end - start) / CLOCKS_PER_SEC;
			printf("forwardsize%d:streamsize%d:%.6lf\n", N, i_stream, fft_time);

			start = clock();
			for (int j = 0; j < test_number / i_stream; ++j) {
				for (int i = 0; i < i_stream; ++i) {
					hipfftExecC2C(plan[i], (hipfftComplex *)d_signal, (hipfftComplex *)d_signal, 
						HIPFFT_BACKWARD);
					hipDeviceSynchronize();
				}
			}
			end = clock();
			fft_time = (double)(end - start) / CLOCKS_PER_SEC;
			printf("backwardsize%d:streamsize%d:%.6lf\n", N, i_stream, fft_time);

			Complex * result[8];
			for (int i = 0; i < i_stream; i++)
				result[i] = new Complex[SIZE];
			for (int i = 0; i < i_stream; i++)
				hipMemcpyAsync(result[i], d_signal[i], sizeof(Complex)*SIZE, 
					hipMemcpyDeviceToHost, streams[i]);

			for (int i = 0; i < i_stream; i++) {
				delete result[i];
				delete fg[i];
			}
			for (int i = 0; i < i_stream; i++)
				hipfftDestroy(plan[i]);
			//hipfftDestroy(plan2);
			for (int i = 0; i < i_stream; i++)
				hipFree(d_signal[i]);
			for (int i = 0; i < i_stream; i++)
				d_signal[i] = NULL;
			for (int i = 0; i < i_stream; i++)
				hipStreamDestroy(streams[i]);
			hipDeviceReset();
		}
	}		
}
